/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <weights.cuh>
#include <mpi.h>
#include <boost/function.hpp>
#include <boost/bind.hpp>

#include "thread.h"
#include "logging.h"

static double _bytesSent = 0;
static double _bytesRecv = 0;
static double _timeWasted = 0;

bool Weights::_autoCopyToGPU = false;
WeightManager* WeightManager::_instance = NULL;

typedef map<int64_t, FreeList<Matrix> > MatrixFL;
static MatrixFL _sendTmp;

class SendBatch {
private:
    Matrix* _delta;
    int64_t _id;
    vector<MPI::Request> _reqs;
public:
    SendBatch(int64_t id, const NVMatrix& delta) :
                    _id(id) {
        TimerBlock tt(_timeWasted);
        _delta = _sendTmp[delta.getNumElements()].get();
        _delta->resize(delta.getNumRows(), delta.getNumCols());
        delta.copyToHost(*_delta);

        Log_Debug("Sending batch... %d", _id);
        for (int i = 0; i < MPI::COMM_WORLD.Get_size(); ++i) {
            if (i == MPI::COMM_WORLD.Get_rank()) {
                continue;
            }
            _reqs.push_back(MPI::COMM_WORLD.Isend(_delta->getData(), _delta->getNumElements(), MPI::FLOAT, i, _id));
            _bytesSent += delta.getNumElements() * 4;
        }
    }

    ~SendBatch() {
        _sendTmp[_delta->getNumElements()].release(_delta);
    }

    bool Finished() {
        return MPI::Request::Testall(_reqs.size(), &_reqs[0]);
    }
};

typedef vector<SendBatch*> OutList;

struct WeightManager::WeightData {
    pthread_mutex_t mutex;
    Matrix inc;
    Matrix tmp;
    OutList outgoing;
    MPI::Request incoming;

    WeightData() {
        pthread_mutex_init(&mutex, NULL);
    }
};

WeightManager* WeightManager::get() {
    if (_instance != NULL) {
        return _instance;
    }

    _instance = new WeightManager();
    return _instance;
}

WeightManager::WeightManager() {
    _recvThread = new FuncThread(boost::bind(&WeightManager::_recvThreadFn, this));
    _sendThread = new FuncThread(boost::bind(&WeightManager::_sendThreadFn, this));
}

#define BEGIN_LOOP_OVER_WEIGHTS\
    for (int i = 0; i < _weights.size(); ++i) {\
                WeightData* w = _weights[i];\
                if (w == NULL) { continue; }\
                ScopedLock l(w->mutex);

#define END_LOOP_OVER_WEIGHTS\
        Sleep(0.001);\
    }

void WeightManager::_recvThreadFn() {
    while (1) {
        BEGIN_LOOP_OVER_WEIGHTS
            MPI::Status stat;
            if (w->incoming.Test(stat)) {
                Log_Info("Receiving for %d", i);
                w->inc.add(w->tmp);
                w->incoming = MPI::COMM_WORLD.Irecv(w->tmp.getData(), w->tmp.getNumElements(), MPI::FLOAT,
                                MPI::ANY_SOURCE, i);
            }

            END_LOOP_OVER_WEIGHTS
    }
}

void WeightManager::_sendThreadFn() {
    while (1) {
        BEGIN_LOOP_OVER_WEIGHTS
            for (OutList::iterator j = w->outgoing.begin(); j != w->outgoing.end();) {
                if ((*j)->Finished()) {
                    Log_Info("Send finished...");
                    delete (*j);
                    j = w->outgoing.erase(j);
                } else {
                    ++j;
                }
            }
            END_LOOP_OVER_WEIGHTS
    }
}

void WeightManager::sendAndRecv(int64_t id, NVMatrix& delta, NVMatrix& weights) {
    Log_Info("Sending update: %d", id);
    if (!_weights[id]) {
        WeightData* w = new WeightData;
        w->tmp.resize(delta.getNumRows(), delta.getNumCols());
        w->inc.resize(delta.getNumRows(), delta.getNumCols());
        // Spin up our first receive for this data.
        w->incoming = MPI::COMM_WORLD.Irecv(w->tmp.getData(), w->tmp.getNumElements(), MPI::FLOAT, MPI::ANY_SOURCE,
                        id);
        _weights[id] = w;
    }

    SendBatch* b = new SendBatch(id, delta);

    WeightData* w = _weights[id];
    {
        ScopedLock l(w->mutex);
        w->outgoing.push_back(b);
    
        weights.add(delta);
        _addTmp.resize(w->inc);
        _addTmp.copyFromHost(w->inc);
        weights.add(_addTmp);
        w->inc.scale(0);
    }
}

int64_t WeightManager::newId() {
    // Just insert an empty slot for now - the actually WeightData variables will be initialized
    // in sendUpdate, when we know the size of matrix to allocate.
    int64_t id = _weights.size();
    Log_Debug("New id: %d", id);
    _weights.push_back(NULL);
    return id;
}
