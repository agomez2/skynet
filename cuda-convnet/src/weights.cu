/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <weights.cuh>
#include <mpi.h>
#include <boost/function.hpp>
#include <boost/bind.hpp>

#include "thread.h"
#include "logging.h"

static double _bytesSent = 0;
static double _bytesRecv = 0;
static double _timeWasted = 0;

bool Weights::_autoCopyToGPU = false;
WeightManager* WeightManager::_instance = NULL;

typedef map<int64_t, FreeList<Matrix> > MatrixFL;
static MatrixFL _sendTmp;

class SendBatch {
private:
    Matrix* _delta;
    int64_t _id;
    vector<MPI::Request> _reqs;
public:
    SendBatch(int64_t id, const NVMatrix& delta) :
                    _id(id) {
        _delta = _sendTmp[delta.getNumElements()].get();
        delta.copyToHost(*_delta);

        TimerBlock tt(_timeWasted);
        Log_Debug("Sending batch... %d", _id);
        for (int i = 0; i < MPI::COMM_WORLD.Get_size(); ++i) {
            if (i == MPI::COMM_WORLD.Get_rank()) {
                continue;
            }
            _reqs.push_back(MPI::COMM_WORLD.Isend(_delta->getData(), _delta->getNumElements(), MPI::FLOAT, i, _id));
            _bytesSent += delta.getNumElements() * 4;
        }
    }

    ~SendBatch() {
        _sendTmp[_delta->getNumElements()].release(_delta);
    }

    bool Finished() {
        return MPI::Request::Testall(_reqs.size(), &_reqs[0]);
    }

    void Wait() {
        TimerBlock tt(_timeWasted);
        MPI::Request::Waitall(_reqs.size(), &_reqs[0]);
        for (int i = 0; i < _reqs.size(); ++i) {
            _reqs[i].Free();
        }
    }
};

typedef vector<SendBatch*> OutList;

struct WeightManager::WeightData {
    pthread_mutex_t mutex;
    NVMatrix inc;
    NVMatrix tmp;
    OutList outgoing;
    MPI::Request incoming;

    WeightData() {
        pthread_mutex_init(&mutex, NULL);
    }
};

WeightManager* WeightManager::get() {
    if (_instance != NULL) {
        return _instance;
    }

    _instance = new WeightManager();
    return _instance;
}

WeightManager::WeightManager() {
    _recvThread = new FuncThread(boost::bind(&WeightManager::_recvThreadFn, this));
    _sendThread = new FuncThread(boost::bind(&WeightManager::_sendThreadFn, this));
}

#define BEGIN_LOOP_OVER_WEIGHTS\
    for (int i = 0; i < _weights.size(); ++i) {\
                WeightData* w = _weights[i];\
                if (!w) { continue; }\
                ScopedLock l(w->mutex);

#define END_LOOP_OVER_WEIGHTS\
        Sleep(0.001);\
    }

void WeightManager::_recvThreadFn() {
    while (1) {
        BEGIN_LOOP_OVER_WEIGHTS
            MPI::Status stat;
            if (w->incoming.Test(stat)) {
                w->inc.add(w->tmp);
                w->incoming.Free();
                w->incoming = MPI::COMM_WORLD.Irecv(w->tmp.getDevData(), w->tmp.getNumElements(), MPI::FLOAT,
                                MPI::ANY_SOURCE, i);
            }

            END_LOOP_OVER_WEIGHTS
    }
}

void WeightManager::_sendThreadFn() {
    while (1) {
        BEGIN_LOOP_OVER_WEIGHTS
            for (OutList::iterator j = w->outgoing.begin(); j != w->outgoing.end();) {
                if ((*j)->Finished()) {
                    delete (*j);
                    j = w->outgoing.erase(j);
                } else {
                    ++j;
                }
            }
            END_LOOP_OVER_WEIGHTS
    }
}

void WeightManager::sendAndRecv(int64_t id, NVMatrix& delta, NVMatrix& weights) {
    Log_Info("Sending update: %d", id);
    if (!_weights[id]) {
        WeightData* w = new WeightData;
        w->tmp.resize(delta.getNumRows(), delta.getNumCols());
        w->inc.resize(delta.getNumRows(), delta.getNumCols());
        // Spin up our first receive for this data.
        w->incoming = MPI::COMM_WORLD.Irecv(w->tmp.getDevData(), w->tmp.getNumElements(), MPI::FLOAT, MPI::ANY_SOURCE,
                        id);
        _weights[id] = w;
    }

    SendBatch* b = new SendBatch(id, delta);

    WeightData* w = _weights[id];
    {
        ScopedLock l(w->mutex);
        w->outgoing.push_back(b);
        weights.add(w->inc);
        w->inc.scale(0);
    }

    weights.add(delta);
}

int64_t WeightManager::newId() {
    // Just insert an empty slot for now - the actually WeightData variables will be initialized
    // in sendUpdate, when we know the size of matrix to allocate.
    int64_t id = _weights.size();
    Log_Debug("New id: %d", id);
    _weights.push_back(NULL);
    return id;
}
