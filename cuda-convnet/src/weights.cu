#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * Copyright (c) 2012, Russell Power (russell.power@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <weights.cuh>
#include <mpi.h>
#include <boost/function.hpp>
#include <boost/bind.hpp>
#include <pthread.h>
#include <math.h>

#include <vector>
#include <map>

#include "thread.h"
#include "logging.h"

bool Weights::_autoCopyToGPU = false;

WeightCombiner::WeightCombiner(double momentum, double decay, double learningRate) :
                momentum(momentum), decay(decay), learningRate(learningRate), numGradients(0), magnitude(0) {
}

void WeightCombiner::newGradient(Matrix& gradient, Matrix& accumulator) {
    if (numGradients == 0) {
      gradient.copy(accumulator);
    } else {
      accumulator.add(gradient);
    }
    magnitude += gradient.norm2();
    ++numGradients;
}

void WeightCombiner::newGradient(NVMatrix& gradient, NVMatrix& accumulator) {
    if (numGradients == 0) {
      gradient.copy(accumulator);
    } else {
      accumulator.add(gradient);
    }
    magnitude += gradient.norm2();
    ++numGradients;
}

void WeightCombiner::apply(NVMatrix& weights, NVMatrix& previous, NVMatrix& grads, int numCases) {
    incTmp.resize(weights);
    grads.scale(learningRate / numCases, incTmp);

    if (momentum > 0) {
        Log_Debug("%.10f - momentum", momentum);
        incTmp.add(previous, momentum);
    }

    if (decay > 0) {
        Log_Debug("%.10f - decay", momentum);
        incTmp.add(weights, -decay * learningRate);
    }
   
    // Log_Info("%f %f %f %f %f %f", learningRate, numCases, momentum, decay, previous.norm2(), grads.norm2());
    weights.add(incTmp);
    incTmp.copy(previous);

    // assert(!isnan(incTmp.norm2()));
    // assert(!isnan(weights.norm2()));
    // assert(weights.norm2() > 0);
    numGradients = 0;
}

Weights::Weights(Weights& srcWeights, float epsW) :
                _srcWeights(&srcWeights), _epsW(epsW), _wc(0), _onGPU(false), _numUpdates(0), _weights(NULL), _weightsInc(
                                NULL), _weightsGrad(NULL) {
    _hWeights = &srcWeights.getCPUW();
    _hWeightsInc = &srcWeights.getCPUWInc();
    _mom = srcWeights.getMom();
    _netMgr = NetworkManager::get();
    _weightId = _netMgr->newId(new WeightCombiner(_mom, _wc, _epsW));
    if (_autoCopyToGPU) {
        copyToGPU();
    }
}

Weights::Weights(Matrix& hWeights, Matrix& hWeightsInc, float epsW, float wc, float mom) :
                _srcWeights(NULL), _hWeights(&hWeights), _hWeightsInc(&hWeightsInc), _numUpdates(0), _epsW(epsW), _wc(
                                wc), _mom(mom), _onGPU(false), _weights(NULL), _weightsInc(NULL), _weightsGrad(NULL) {
    _netMgr = NetworkManager::get();
    _weightId = _netMgr->newId(new WeightCombiner(_mom, _wc, _epsW));
    if (_autoCopyToGPU) {
        copyToGPU();
    }
}

Weights::~Weights() {
    delete _hWeights;
    delete _hWeightsInc;
    if (_srcWeights == NULL) {
        delete _weights;
        delete _weightsInc;
        delete _weightsGrad;
    }
}

void Weights::copyToGPU() {
    if (_srcWeights == NULL) {
        _weights = new NVMatrix();
        _weightsInc = new NVMatrix();
        _weightsGrad = new NVMatrix();
        _weights->copyFromHost(*_hWeights, true);
        _weightsInc->copyFromHost(*_hWeightsInc, true);
        _weightsGrad->copyFromHost(*_hWeightsInc, true);
    } else {
        _weights = _srcWeights->_weights;
        _weightsInc = _srcWeights->_weightsInc;
        _weightsGrad = _srcWeights->_weightsGrad;
    }
    _onGPU = true;

//    Log_Debug("%f %f", _weights->norm2(), _weightsInc->norm2());
}

void Weights::update(int numCases) {
    // Only true owner of weights updates
    if (_srcWeights == NULL && _epsW > 0) {
        assert(_onGPU);

        _netMgr->sendAndRecv(_weightId, *_weightsGrad, *_weightsInc, *_weights, numCases);
        _weightsGrad->scale(0);
//        assert(!isnan(_weightsGrad->norm2()));
        _numUpdates = 0;
    } else {
      // Log_Info("Skipping update...");
    }
}

NetworkManager* NetworkManager::_instance = NULL;

using namespace std;

typedef map<int64_t, FreeList<Matrix> > MatrixFL;

// We use 2 weight vectors to send out data, 'sending' and 'pending'.
// The network thread attempts to push data from 'sending' as fast
// as possible, whether or not updates have been created.
//
// Whenever a new delta is produced, we update the 'pending' vector.
//
// As soon as a batch of updates is sent, the 'sending' and 'pending'
// vectors are swapped.
class OutgoingWeights {
private:
    Matrix *_sending;
    Matrix *_pending;
    Matrix *_tmp;
    int64_t _id;

    vector<MPI::Request> _reqs;
public:
    OutgoingWeights(int64_t id, int numRows, int numCols) :
                    _id(id) {
        _sending = new Matrix(numRows, numCols);
        _pending = new Matrix(numRows, numCols);
        _tmp = new Matrix(numRows, numCols);
    }

    void addDelta(const NVMatrix& m) {
        m.copyToHost(*_tmp);
        _pending->add(*_tmp);
    }

    void startSend() {
        for (int i = 0; i < MPI::COMM_WORLD.Get_size(); ++i) {
            if (i == MPI::COMM_WORLD.Get_rank()) {
                continue;
            }
            // Log_Debug("Sending batch... %d %d", _id, _out->getNumElements() * 4);
            _reqs.push_back(MPI::COMM_WORLD.Isend(_sending->getData(), _sending->getNumElements(), MPI::FLOAT, i, _id));
        }
    }

    bool sendDone() {
        return _reqs.empty() || MPI::Request::Testall(_reqs.size(), &_reqs[0]);
    }

    void swapPending() {
        _sending->scale(0);
        std::swap(_sending, _pending);
        _reqs.clear();
    }
};

class IncomingWeights {
private:
    MPI::Request _req;
    bool _started;
    int64_t _id;
    Matrix *_tgt;
public:
    IncomingWeights(int64_t id, Matrix* tgt) :
                    _started(false), _id(id), _tgt(tgt) {
    }

    void startRecv() {
        assert(!_started);
        _started = true;
        _req = MPI::COMM_WORLD.Irecv(_tgt->getData(), _tgt->getNumElements(), MPI::FLOAT, MPI::ANY_SOURCE, _id);
    }

    bool recvDone() {
        MPI::Status stat;
        bool done = _req.Test(stat);
        if (!done) {
            return false;
        }
        Log_Assert(stat.Get_count(MPI::FLOAT) == _tgt->getNumElements(), "Unexpected recv: %d %d %d",
                        _id, _tgt->getNumElements() * 4, stat.Get_count(MPI::FLOAT));
        return true;
    }

    void reset() {
        _started = false;
    }
};

WeightData::WeightData(int64_t id, WeightCombiner* combiner) {
    pthread_mutex_init(&sendMutex, NULL);
    pthread_mutex_init(&recvMutex, NULL);

    incReady = false;
    incCount = 0;
    this->id = id;

    incoming = NULL;
    outgoing = NULL;
    this->combiner = combiner;

    initialized = false;
}

void WeightData::initialize(int numRows, int numCols) {
    outgoing = new OutgoingWeights(id, numRows, numCols);
    recvTmp.resize(numRows, numCols);
    inc.resize(numRows, numCols);
    inc.scale(0);
    initialized = true;
}

bool WeightData::handleRecv() {
    if (incoming == NULL) {
        incoming = new IncomingWeights(id, &recvTmp);
        incoming->startRecv();
    }
    if (incoming->recvDone()) {
        {
            ScopedLock l(recvMutex);
            combiner->newGradient(recvTmp, inc);
            incReady = true;
        }
        incoming->reset();
        incoming->startRecv();
        return true;
    }
    return false;
}

bool WeightData::handleSend() {
    {
        ScopedLock l(sendMutex);
        if (outgoing->sendDone()) {
            outgoing->swapPending();
            outgoing->startSend();
            return true;
        }
        return false;
    }
}

NetworkManager* NetworkManager::get() {
    if (_instance != NULL) {
        return _instance;
    }

    _instance = new NetworkManager();
    return _instance;
}

NetworkManager::NetworkManager() {
    _cudaDevice = -1;
    _pause = _isPaused = false;
    _mpiThread = NULL;
    _bytesRecv = 0;
    _bytesSent = 0;
    _timeWasted = 0;
}

void NetworkManager::initialize() {
    NetworkManager* w = NetworkManager::get();
    assert(hipGetDevice(&w->_cudaDevice) == hipSuccess);
    w->_mpiThread = new FuncThread(boost::bind(&NetworkManager::_mpiThreadFn, w));
}

void NetworkManager::pauseMPI() {
    NetworkManager::get()->_pause = true;
    while (!NetworkManager::get()->_isPaused) {
        Sleep(0.001);
    }

    Log_Debug("MPI thread paused.");
}

void NetworkManager::resumeMPI() {
    NetworkManager::get()->_pause = false;
    while (NetworkManager::get()->_isPaused) {
        Sleep(0.001);
    }

    Log_Debug("MPI thread resumed.");
}

void NetworkManager::_mpiThreadFn() {
    Log_Debug("Starting MPI worker thread, using CUDA device: %d", _cudaDevice);
    assert(hipSetDevice(_cudaDevice) == hipSuccess);
    cublasInit();
    while (1) {
        Sleep(0.01);
        if (_pause) {
            _isPaused = true;
            continue;
        }

        _isPaused = false;
        for (int i = 0; i < _weights.size(); ++i) {
            WeightData* w = _weights[i];
            if (!w->initialized) {
                continue;
            }

            if (w->handleRecv()) {
                _bytesRecv += w->recvTmp.getNumDataBytes();
            }
            if (w->handleSend()) {
                _bytesSent += w->recvTmp.getNumDataBytes() * (MPI::COMM_WORLD.Get_size() - 1);
            }
        }
    }
}

void NetworkManager::sendAndRecv(int64_t id, NVMatrix& gradient, NVMatrix& increment, NVMatrix& weights, int numCases) {
    TimerBlock tt(_timeWasted);

    WeightData* w = _weights[id];
    w->combiner->transformGradient(gradient);
    
    if (!w->initialized) {
        ScopedLock lw(w->sendMutex);
        ScopedLock lr(w->recvMutex);
        w->initialize(gradient.getNumRows(), gradient.getNumCols());
    }

    if (MPI::COMM_WORLD.Get_size() > 1) {
        ScopedLock l(w->sendMutex);
        w->outgoing->addDelta(gradient);
    }

    if (w->incReady) {
        ScopedLock l(w->recvMutex);
        assert(gradient.getNumRows() == w->inc.getNumRows());
        assert(gradient.getNumCols() == w->inc.getNumCols());

        _gpuTmp.resize(w->inc);
        _gpuTmp.copyFromHost(w->inc);
        w->combiner->newGradient(gradient, _gpuTmp);
        w->combiner->apply(weights, increment, _gpuTmp, numCases);
        w->inc.scale(0);
        w->incCount = 0;
    } else {
        _gpuTmp.resize(gradient);
        w->combiner->newGradient(gradient, _gpuTmp);
        w->combiner->apply(weights, increment, _gpuTmp, numCases);
    }

    // PERIODIC(30, Log_Debug("MPI status: %.2fMB sent, %.2fMB received, %.2f seconds", _bytesSent / 1e6, _bytesRecv / 1e6, _timeWasted));
}

int64_t NetworkManager::newId(WeightCombiner* combiner) {
    int64_t id = _weights.size();
    _weights.push_back(new WeightData(id, combiner));
    return id;
}
