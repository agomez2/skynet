/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <weights.cuh>
#include <mpi.h>
#include <boost/function.hpp>
#include <boost/bind.hpp>

#include "thread.h"
#include "logging.h"

static int64_t _bytesSent = 0;
static int64_t _bytesRecv = 0;
static double _timeWasted = 0;

bool Weights::_autoCopyToGPU = false;
WeightManager* WeightManager::_instance = NULL;

typedef map<int64_t, FreeList<Matrix> > MatrixFL;
static MatrixFL _sendTmp;

class OutgoingWeights {
private:
    Matrix* _delta;
    int64_t _id;
    bool _sent;
    vector<MPI::Request> _reqs;
public:
    OutgoingWeights(int64_t id, const NVMatrix& delta) :
                    _id(id), _sent(false) {
        TimerBlock tt(_timeWasted);
        _delta = _sendTmp[delta.getNumElements()].get();
        _delta->resize(delta.getNumRows(), delta.getNumCols());
        delta.copyToHost(*_delta);
    }

    ~OutgoingWeights() {
        _sendTmp[_delta->getNumElements()].release(_delta);
    }

    bool getSent() { return _sent; }

    void Send() {
        assert(!_sent);
        _sent = true;
        for (int i = 0; i < MPI::COMM_WORLD.Get_size(); ++i) {
            if (i == MPI::COMM_WORLD.Get_rank()) {
                continue;
            }
            // Log_Info("Sending batch... %d %d", _id, _delta->getNumElements() * 4);
            _reqs.push_back(MPI::COMM_WORLD.Isend(_delta->getData(), _delta->getNumElements(), MPI::FLOAT, i, _id));
            _bytesSent += _delta->getNumElements() * 4;
        }
    }

    bool Finished() {
        return MPI::Request::Testall(_reqs.size(), &_reqs[0]);
    }
};

class IncomingWeights {
private:
    MPI::Request _req;
    bool _started;
    int64_t _id;
    Matrix *_tgt;
public:
    IncomingWeights(int64_t id, Matrix* tgt) : _started(false), _id(id), _tgt(tgt) {
    }

    void StartRecv() {
      assert(!_started);
      _started = true;
      _req = MPI::COMM_WORLD.Irecv(_tgt->getData(), _tgt->getNumElements(), MPI::FLOAT, MPI::ANY_SOURCE, _id);
    }

    bool Finished() {
      MPI::Status stat;
      bool done = _req.Test(stat);
      if (!done) { return false; }
      Log_Assert(stat.Get_count(MPI::FLOAT) == _tgt->getNumElements(),
                 "Unexpected recv: %d %d %d", _id, _tgt->getNumElements() * 4, stat.Get_count(MPI::FLOAT));
      return true;
    }

    void Reset() {
      _started = false;
    }
};

typedef vector<OutgoingWeights*> OutList;

struct WeightManager::WeightData {
    pthread_mutex_t sendMutex;
    pthread_mutex_t recvMutex;

    Matrix inc;
    Matrix tmp;
    OutList outgoing;
    IncomingWeights* incoming;

    WeightData() {
        pthread_mutex_init(&sendMutex, NULL);
        pthread_mutex_init(&recvMutex, NULL);
    }
};

WeightManager* WeightManager::get() {
    if (_instance != NULL) {
        return _instance;
    }

    _instance = new WeightManager();
    return _instance;
}

WeightManager::WeightManager() {
    // _recvThread = new FuncThread(boost::bind(&WeightManager::_recvThreadFn, this));
    // _sendThread = new FuncThread(boost::bind(&WeightManager::_sendThreadFn, this));
    _mpiThread = new FuncThread(boost::bind(&WeightManager::_mpiThreadFn, this));
}


void WeightManager::_mpiThreadFn() {
    while (1) {
        Sleep(0.001);
        for (int i = 0; i < _weights.size(); ++i) {
            WeightData* w = _weights[i];
            if (w == NULL) { 
                continue; 
            }

            // check for and receive incoming data...
            if (w->incoming == NULL) {
                w->incoming = new IncomingWeights(i, &w->tmp);
                w->incoming->StartRecv();
            }

            if (w->incoming->Finished()) {
                ScopedLock l(w->recvMutex);
                w->inc.add(w->tmp);
                w->incoming->Reset();
                w->incoming->StartRecv();
                _bytesRecv += w->tmp.getNumElements() * 4;
            }

            {
              ScopedLock l(w->sendMutex);
              for (OutList::iterator j = w->outgoing.begin(); j != w->outgoing.end();) {
                  OutgoingWeights* o = *j;
                  if (!o->getSent()) { o->Send(); }
                  if (o->Finished()) {
                      delete o;
                      j = w->outgoing.erase(j);
                  } else {
                      ++j;
                  }
              }
           }
        }
    }
}


void WeightManager::sendAndRecv(int64_t id, NVMatrix& delta, NVMatrix& weights) {
    weights.add(delta);

    TimerBlock tt(_timeWasted);
    if (!_weights[id]) {
        Log_Info("New weight vector %d - %d", id, delta.getNumElements() * 4);
        WeightData* w = new WeightData;
        w->tmp.resize(delta.getNumRows(), delta.getNumCols());
        w->inc.resize(delta.getNumRows(), delta.getNumCols());
        w->incoming = NULL;
        _weights[id] = w;
    }

    OutgoingWeights* b = new OutgoingWeights(id, delta);
    WeightData* w = _weights[id];
    {
        ScopedLock l(w->sendMutex);
        w->outgoing.push_back(b);
    }

    {
        ScopedLock l(w->recvMutex);
        assert(delta.getNumRows() == w->tmp.getNumRows());
        assert(delta.getNumCols() == w->tmp.getNumCols());

        _addTmp.resize(w->inc);
        _addTmp.copyFromHost(w->inc);
        weights.add(_addTmp);
        //weights.add(w->inc);
        w->inc.scale(0);
    }

    PERIODIC(5, Log_Info("MPI status: %.2fMB sent, %.2fMB received, %.2f seconds", _bytesSent / 1e6, _bytesRecv / 1e6, _timeWasted));
}

int64_t WeightManager::newId() {
    // Just insert an empty slot for now - the actually WeightData variables will be initialized
    // in sendUpdate, when we know the size of matrix to allocate.
    int64_t id = _weights.size();
    Log_Debug("New id: %d", id);
    _weights.push_back(NULL);
    return id;
}
