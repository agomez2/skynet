/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * Copyright (c) 2012, Russell Power (russell.power@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <weights.cuh>
#include <mpi.h>
#include <boost/function.hpp>
#include <boost/bind.hpp>
#include <pthread.h>

#include <vector>
#include <map>

#include "thread.h"
#include "logging.h"

bool Weights::_autoCopyToGPU = false;

Weights::Weights(Weights& srcWeights, float epsW) :
                _srcWeights(&srcWeights), _epsW(epsW), _wc(0), _onGPU(false), _numUpdates(0), _weights(NULL), _weightsInc(
                                NULL), _weightsGrad(NULL) {
    _hWeights = &srcWeights.getCPUW();
    _hWeightsInc = &srcWeights.getCPUWInc();
    _mom = srcWeights.getMom();
    _netMgr = NetworkManager::get();
    _weightId = _netMgr->newId();
    if (_autoCopyToGPU) {
        copyToGPU();
    }
}

Weights::Weights(Matrix& hWeights, Matrix& hWeightsInc, float epsW, float wc, float mom) :
                _srcWeights(NULL), _hWeights(&hWeights), _hWeightsInc(&hWeightsInc), _numUpdates(0), _epsW(epsW), _wc(
                                wc), _mom(mom), _onGPU(false), _weights(NULL), _weightsInc(NULL), _weightsGrad(NULL) {
    _netMgr = NetworkManager::get();
    _weightId = _netMgr->newId();
    if (_autoCopyToGPU) {
        copyToGPU();
    }
}

Weights::~Weights() {
    delete _hWeights;
    delete _hWeightsInc;
    if (_srcWeights == NULL) {
        delete _weights;
        delete _weightsInc;
        delete _weightsGrad;
    }
}

void Weights::copyToGPU() {
    if (_srcWeights == NULL) {
        _weights = new NVMatrix();
        _weightsInc = new NVMatrix();
        _weightsGrad = new NVMatrix();
        _weights->copyFromHost(*_hWeights, true);
        _weightsInc->copyFromHost(*_hWeightsInc, true);
//        _weightsGrad->resize(_weightsInc->getNumRows(), _weightsInc->getNumCols());
        _weightsGrad->resize(*_weightsInc);
        Log_Info("Gradients resized to %d %d", _weightsGrad->getNumRows(), _weightsGrad->getNumCols());
        Log_Info("Weights are sized: %d %d", _weightsInc->getNumRows(), _weightsInc->getNumCols());
    } else {
        _weights = _srcWeights->_weights;
        _weightsInc = _srcWeights->_weightsInc;
        _weightsGrad = _srcWeights->_weightsGrad;
    }
    _onGPU = true;
}

void Weights::update(int numCases) {
    // Only true owner of weights updates
    if (_srcWeights == NULL && _epsW > 0) {
        assert(_onGPU);

        _weightsInc->scale(0);
        _weightsInc->add(*_weightsInc, _mom);
        _weightsInc->add(*_weightsGrad, _epsW / numCases);
        if (_wc > 0) {
            _weightsInc->add(*_weights, -_wc * _epsW);
        }

        _netMgr->sendAndRecv(_weightId, *_weightsInc, *_weights);
        _numUpdates = 0;
    }
}

NetworkManager* NetworkManager::_instance = NULL;

using namespace std;

typedef map<int64_t, FreeList<Matrix> > MatrixFL;

// We use 2 weight vectors to send out data, 'sending' and 'pending'.
// The network thread attempts to push data from 'sending' as fast
// as possible, whether or not updates have been created.
//
// Whenever a new delta is produced, we update the 'pending' vector.
//
// As soon as a batch of updates is sent, the 'sending' and 'pending'
// vectors are swapped.
class OutgoingWeights {
private:
    Matrix *_sending;
    Matrix *_pending;
    Matrix *_tmp;
    int64_t _id;

    vector<MPI::Request> _reqs;
public:
    OutgoingWeights(int64_t id, int numRows, int numCols) :
                    _id(id) {
        _sending = new Matrix(numRows, numCols);
        _pending = new Matrix(numRows, numCols);
        _tmp = new Matrix(numRows, numCols);
    }

    void addDelta(const NVMatrix& m) {
        m.copyToHost(*_tmp);
        _pending->add(*_tmp);
    }

    void startSend() {
        for (int i = 0; i < MPI::COMM_WORLD.Get_size(); ++i) {
            if (i == MPI::COMM_WORLD.Get_rank()) {
                continue;
            }
            // Log_Info("Sending batch... %d %d", _id, _out->getNumElements() * 4);
            _reqs.push_back(MPI::COMM_WORLD.Isend(_sending->getData(), _sending->getNumElements(), MPI::FLOAT, i, _id));
        }
    }

    bool sendDone() {
        return _reqs.empty() || MPI::Request::Testall(_reqs.size(), &_reqs[0]);
    }

    void swapPending() {
        _sending->scale(0);
        std::swap(_sending, _pending);
        _reqs.clear();
    }
};

class IncomingWeights {
private:
    MPI::Request _req;
    bool _started;
    int64_t _id;
    Matrix *_tgt;
public:
    IncomingWeights(int64_t id, Matrix* tgt) :
                    _started(false), _id(id), _tgt(tgt) {
    }

    void startRecv() {
        assert(!_started);
        _started = true;
        _req = MPI::COMM_WORLD.Irecv(_tgt->getData(), _tgt->getNumElements(), MPI::FLOAT, MPI::ANY_SOURCE, _id);
    }

    bool recvDone() {
        MPI::Status stat;
        bool done = _req.Test(stat);
        if (!done) {
            return false;
        }
        Log_Assert(stat.Get_count(MPI::FLOAT) == _tgt->getNumElements(), "Unexpected recv: %d %d %d",
                        _id, _tgt->getNumElements() * 4, stat.Get_count(MPI::FLOAT));
        return true;
    }

    void reset() {
        _started = false;
    }
};

WeightData::WeightData(int64_t id, int numRows, int numCols) {
    pthread_mutex_init(&sendMutex, NULL);
    pthread_mutex_init(&recvMutex, NULL);
    outgoing = new OutgoingWeights(id, numRows, numCols);
    this->id = id;

    recvTmp.resize(numRows, numCols);
    inc.resize(numRows, numCols);
    incReady = false;
    incoming = NULL;
    incCount = 0;
}

bool WeightData::handleRecv() {
    if (incoming == NULL) {
        incoming = new IncomingWeights(id, &recvTmp);
        incoming->startRecv();
    }
    if (incoming->recvDone()) {
        {
            // _gpuTmp is shared across WeightData instances, but is only used from the MPI thread.
            // _gpuTmp.resize(inc);
            // _gpuTmp.copyFromHost(recvTmp);
            ScopedLock l(recvMutex);
            // inc.add(_gpuTmp);
            inc.add(recvTmp);
            incReady = true;
        }
        incoming->reset();
        incoming->startRecv();
        return true;
    }
    return false;
}

bool WeightData::handleSend() {
    {
        ScopedLock l(sendMutex);
        if (outgoing->sendDone()) {
            outgoing->swapPending();
            outgoing->startSend();
            return true;
        }
        return false;
    }
}

NetworkManager* NetworkManager::get() {
    if (_instance != NULL) {
        return _instance;
    }

    _instance = new NetworkManager();
    return _instance;
}

NetworkManager::NetworkManager() {
    _cudaDevice = -1;
    _pause = _isPaused = false;
    _mpiThread = NULL;
    _bytesRecv = 0;
    _bytesSent = 0;
    _timeWasted = 0;
}

void NetworkManager::initialize() {
    NetworkManager* w = NetworkManager::get();
    assert(hipGetDevice(&w->_cudaDevice) == hipSuccess);
    w->_mpiThread = new FuncThread(boost::bind(&NetworkManager::_mpiThreadFn, w));
}

void NetworkManager::pauseMPI() {
    NetworkManager::get()->_pause = true;
    while (!NetworkManager::get()->_isPaused) {
        Sleep(0.001);
    }

    Log_Debug("MPI thread paused.");
}

void NetworkManager::resumeMPI() {
    NetworkManager::get()->_pause = false;
    while (NetworkManager::get()->_isPaused) {
        Sleep(0.001);
    }

    Log_Debug("MPI thread resumed.");
}

void NetworkManager::_mpiThreadFn() {
    Log_Info("Starting MPI worker thread, using CUDA device: %d", _cudaDevice);
    assert(hipSetDevice(_cudaDevice) == hipSuccess);
    cublasInit();
    while (1) {
        Sleep(0.01);
        if (_pause) {
            _isPaused = true;
            continue;
        }

        _isPaused = false;
        for (int i = 0; i < _weights.size(); ++i) {
            WeightData* w = _weights[i];
            if (w == NULL) {
                continue;
            }

            if (w->handleRecv()) {
                _bytesRecv += w->recvTmp.getNumDataBytes();
            }
            if (w->handleSend()) {
                _bytesSent += w->recvTmp.getNumDataBytes() * (MPI::COMM_WORLD.Get_size() - 1);
            }
        }
    }
}

void NetworkManager::sendAndRecv(int64_t id, NVMatrix& delta, NVMatrix& weights) {
    weights.add(delta);

    TimerBlock tt(_timeWasted);
    if (!_weights[id]) {
        Log_Info("New weight vector %d - %d", id, delta.getNumElements() * 4);
        WeightData* w = new WeightData(id, delta.getNumRows(), delta.getNumCols());
        _weights[id] = w;
    }

    WeightData* w = _weights[id];
    {
        ScopedLock l(w->sendMutex);
        w->outgoing->addDelta(delta);
    }

    if (w->incReady) {
        ScopedLock l(w->recvMutex);
        assert(delta.getNumRows() == w->inc.getNumRows());
        assert(delta.getNumCols() == w->inc.getNumCols());

        _gpuTmp.resize(w->inc);
        _gpuTmp.copyFromHost(w->inc);
        _gpuTmp.add(delta);
        weights.add(_gpuTmp, 1 / (1.0 + w->incCount));

        // weights.add(_gpuTmp);

        // w->inc.add(delta);
        // weights.add(w->inc);
        w->inc.scale(0);
        w->incCount = 0;
    } else {
        weights.add(delta);
    }

    PERIODIC(5,
                    Log_Info("MPI status: %.2fMB sent, %.2fMB received, %.2f seconds", _bytesSent / 1e6, _bytesRecv / 1e6, _timeWasted));
}

int64_t NetworkManager::newId() {
    // Just insert an empty slot for now - the actually WeightData variables will be initialized
    // in sendUpdate, when we know the size of matrix to allocate.
    int64_t id = _weights.size();
    Log_Debug("New id: %d", id);
    _weights.push_back(NULL);
    return id;
}
