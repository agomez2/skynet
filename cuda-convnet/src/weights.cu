/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <weights.cuh>
#include <mpi.h>
#include <boost/function.hpp>
#include <boost/bind.hpp>

#include "thread.h"
#include "logging.h"

static int64_t _bytesSent = 0;
static int64_t _bytesRecv = 0;
static double _timeWasted = 0;

bool Weights::_autoCopyToGPU = false;
WeightManager* WeightManager::_instance = NULL;

typedef map<int64_t, FreeList<Matrix> > MatrixFL;

class OutgoingWeights {
private:
    Matrix *_sending;
    Matrix *_pending;
    Matrix *_tmp;
    int64_t _id;
    
    vector<MPI::Request> _reqs;
public:
    OutgoingWeights(int64_t id, int numRows, int numCols) : _id(id) {
        _sending = new Matrix(numRows, numCols);
        _pending = new Matrix(numRows, numCols);
        _tmp = new Matrix(numRows, numCols);
    }

    void Add(const NVMatrix& m) {
      m.copyToHost(*_tmp);
      _pending->add(*_tmp);
    }

    void Send() {
        for (int i = 0; i < MPI::COMM_WORLD.Get_size(); ++i) {
            if (i == MPI::COMM_WORLD.Get_rank()) {
                continue;
            }
            // Log_Info("Sending batch... %d %d", _id, _out->getNumElements() * 4);
            _reqs.push_back(MPI::COMM_WORLD.Isend(_sending->getData(), _sending->getNumElements(), MPI::FLOAT, i, _id));
            _bytesSent += _sending->getNumElements() * 4;
        }
    }

    bool Finished() {
        return _reqs.empty() || MPI::Request::Testall(_reqs.size(), &_reqs[0]);
    }

    void swapPending() {
        _sending->scale(0);
        std::swap(_sending, _pending);
        _reqs.clear();
    }
};

class IncomingWeights {
private:
    MPI::Request _req;
    bool _started;
    int64_t _id;
    Matrix *_tgt;
public:
    IncomingWeights(int64_t id, Matrix* tgt) : _started(false), _id(id), _tgt(tgt) {
    }

    void StartRecv() {
      assert(!_started);
      _started = true;
      _req = MPI::COMM_WORLD.Irecv(_tgt->getData(), _tgt->getNumElements(), MPI::FLOAT, MPI::ANY_SOURCE, _id);
    }

    bool Finished() {
      MPI::Status stat;
      bool done = _req.Test(stat);
      if (!done) { return false; }
      Log_Assert(stat.Get_count(MPI::FLOAT) == _tgt->getNumElements(),
                 "Unexpected recv: %d %d %d", _id, _tgt->getNumElements() * 4, stat.Get_count(MPI::FLOAT));
      return true;
    }

    void Reset() {
      _started = false;
    }
};

static NVMatrix _gpuTmp = NULL;

struct WeightManager::WeightData {
    pthread_mutex_t sendMutex;
    pthread_mutex_t recvMutex;

    // NVMatrix inc;
    Matrix inc;
    bool incReady;

    Matrix recvTmp;
    OutgoingWeights* outgoing;
    IncomingWeights* incoming;

    int64_t id;

    WeightData(int64_t id, int numRows, int numCols) {
        pthread_mutex_init(&sendMutex, NULL);
        pthread_mutex_init(&recvMutex, NULL);
        recvTmp.resize(numRows, numCols);
        inc.resize(numRows, numCols);
        incoming = NULL;
        outgoing = new OutgoingWeights(id, numRows, numCols);

        this->id = id;
        incReady = false;
    }

    void handleRecv() {
        if (incoming == NULL) {
            incoming = new IncomingWeights(id, &recvTmp);
            incoming->StartRecv();
        }

        if (incoming->Finished()) {
            {
                // _gpuTmp is shared across WeightData instances, but is only used from the MPI thread.
                // _gpuTmp.resize(inc);
                // _gpuTmp.copyFromHost(recvTmp);
                
                ScopedLock l(recvMutex);
                // inc.add(_gpuTmp);
                inc.add(recvTmp);
                incReady = true;
            }
            incoming->Reset();
            incoming->StartRecv();
            _bytesRecv += recvTmp.getNumElements() * 4;
        }
    }

    void handleSend() {
        {
          ScopedLock l(sendMutex);
          if (outgoing->Finished()) {
            outgoing->swapPending();
            outgoing->Send();
          }
       }
    }
};

WeightManager* WeightManager::get() {
    if (_instance != NULL) {
        return _instance;
    }

    _instance = new WeightManager();
    return _instance;
}

WeightManager::WeightManager() {
    // _recvThread = new FuncThread(boost::bind(&WeightManager::_recvThreadFn, this));
    // _sendThread = new FuncThread(boost::bind(&WeightManager::_sendThreadFn, this));
}

void WeightManager::initialize() {
    WeightManager* w = WeightManager::get();
    assert(hipGetDevice(&w->_cudaDevice) == hipSuccess);
    w->_mpiThread = new FuncThread(boost::bind(&WeightManager::_mpiThreadFn, w));
}


void WeightManager::_mpiThreadFn() {
    Log_Info("Starting MPI worker thread, using CUDA device: %d", _cudaDevice);
    assert(hipSetDevice(_cudaDevice) == hipSuccess);
    cublasInit();
    while (1) {
        Sleep(0.01);
        for (int i = 0; i < _weights.size(); ++i) {
            WeightData* w = _weights[i];
            if (w == NULL) { 
                continue; 
            }

            w->handleRecv();
            w->handleSend();
        }
    }
}


void WeightManager::sendAndRecv(int64_t id, NVMatrix& delta, NVMatrix& weights) {
    weights.add(delta);

    TimerBlock tt(_timeWasted);
    if (!_weights[id]) {
        Log_Info("New weight vector %d - %d", id, delta.getNumElements() * 4);
        WeightData* w = new WeightData(id, delta.getNumRows(), delta.getNumCols());
        _weights[id] = w;
    }

    WeightData* w = _weights[id];
    {
        ScopedLock l(w->sendMutex);
        w->outgoing->Add(delta);
    }

    if (w->incReady) {
        ScopedLock l(w->recvMutex);
        assert(delta.getNumRows() == w->inc.getNumRows());
        assert(delta.getNumCols() == w->inc.getNumCols());

        _gpuTmp.resize(w->inc);
        _gpuTmp.copyFromHost(w->inc);
        weights.add(_gpuTmp);
        // weights.add(w->inc);
        w->inc.scale(0);
    }

    PERIODIC(5, Log_Info("MPI status: %.2fMB sent, %.2fMB received, %.2f seconds", _bytesSent / 1e6, _bytesRecv / 1e6, _timeWasted));
}

int64_t WeightManager::newId() {
    // Just insert an empty slot for now - the actually WeightData variables will be initialized
    // in sendUpdate, when we know the size of matrix to allocate.
    int64_t id = _weights.size();
    Log_Debug("New id: %d", id);
    _weights.push_back(NULL);
    return id;
}
