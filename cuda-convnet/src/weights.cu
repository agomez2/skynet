#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * Copyright (c) 2012, Russell Power (russell.power@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <weights.cuh>
#include <mpi.h>
#include <boost/function.hpp>
#include <boost/bind.hpp>
#include <pthread.h>
#include <math.h>

#include <vector>
#include <map>

#include "thread.h"
#include "logging.h"

bool Weights::_autoCopyToGPU = false;

WeightCombiner::WeightCombiner(double momentum, double decay, double learningRate) :
                momentum(momentum), decay(decay), learningRate(learningRate), numGradients(0) {
}

void WeightCombiner::newGradient(Matrix& gradient, Matrix& accumulator) {
    accumulator.add(gradient);
    ++numGradients;
}

void WeightCombiner::newGradient(NVMatrix& gradient, NVMatrix& accumulator) {
    accumulator.add(gradient);
    ++numGradients;
}

void WeightCombiner::apply(NVMatrix& weights, NVMatrix& previous, NVMatrix& grads, int numCases) {
    incTmp.resize(weights);
    incTmp.scale(0);
    incTmp.add(previous, momentum);
    incTmp.add(grads, learningRate / numCases);
    if (decay > 0) {
        incTmp.add(weights, -decay * learningRate);
    }
    previous.copy(incTmp);
    weights.add(incTmp);
}

void AdagradCombiner::newGradient(Matrix& gradient, Matrix& accumulator) {
    WeightCombiner::newGradient(gradient, accumulator);
    _magnitude += gradient.norm2();
}

void AdagradCombiner::newGradient(NVMatrix& gradient, NVMatrix& accumulator) {
    WeightCombiner::newGradient(gradient, accumulator);
    _magnitude += gradient.norm2();
}

void AdagradCombiner::apply(NVMatrix& weights, NVMatrix& previous, NVMatrix& grads, int numCases) {
    incTmp.resize(weights);
    incTmp.scale(0);
    //        incTmp.add(incTmp, momentum);
    double adaptiveRate = learningRate / sqrt(_magnitude);
    incTmp.add(grads, adaptiveRate / numCases);
    if (decay > 0) {
        incTmp.add(weights, -decay * learningRate);
    }
    previous.copy(incTmp);
    weights.add(incTmp);
}

Weights::Weights(Weights& srcWeights, float epsW) :
                _srcWeights(&srcWeights), _epsW(epsW), _wc(0), _onGPU(false), _numUpdates(0), _weights(NULL), _weightsInc(
                                NULL), _weightsGrad(NULL) {
    _hWeights = &srcWeights.getCPUW();
    _hWeightsInc = &srcWeights.getCPUWInc();
    _mom = srcWeights.getMom();
    _netMgr = NetworkManager::get();
    _weightId = _netMgr->newId(new AdagradCombiner(_mom, _wc, _epsW));
    if (_autoCopyToGPU) {
        copyToGPU();
    }
}

Weights::Weights(Matrix& hWeights, Matrix& hWeightsInc, float epsW, float wc, float mom) :
                _srcWeights(NULL), _hWeights(&hWeights), _hWeightsInc(&hWeightsInc), _numUpdates(0), _epsW(epsW), _wc(
                                wc), _mom(mom), _onGPU(false), _weights(NULL), _weightsInc(NULL), _weightsGrad(NULL) {
    _netMgr = NetworkManager::get();
    _weightId = _netMgr->newId(new AdagradCombiner(_mom, _wc, _epsW));
    if (_autoCopyToGPU) {
        copyToGPU();
    }
}

Weights::~Weights() {
    delete _hWeights;
    delete _hWeightsInc;
    if (_srcWeights == NULL) {
        delete _weights;
        delete _weightsInc;
        delete _weightsGrad;
    }
}

void Weights::copyToGPU() {
    if (_srcWeights == NULL) {
        _weights = new NVMatrix();
        _weightsInc = new NVMatrix();
        _weightsGrad = new NVMatrix();
        _weights->copyFromHost(*_hWeights, true);
        _weightsInc->copyFromHost(*_hWeightsInc, true);
//        _weightsGrad->resize(_weightsInc->getNumRows(), _weightsInc->getNumCols());
        _weightsGrad->resize(*_weightsInc);
    } else {
        _weights = _srcWeights->_weights;
        _weightsInc = _srcWeights->_weightsInc;
        _weightsGrad = _srcWeights->_weightsGrad;
    }
    _onGPU = true;
}

void Weights::update(int numCases) {
    // Only true owner of weights updates
    if (_srcWeights == NULL && _epsW > 0) {
        assert(_onGPU);

        _netMgr->sendAndRecv(_weightId, *_weightsGrad, *_weightsInc, *_weights, numCases);
        _numUpdates = 0;
    }
}

NetworkManager* NetworkManager::_instance = NULL;

using namespace std;

typedef map<int64_t, FreeList<Matrix> > MatrixFL;

// We use 2 weight vectors to send out data, 'sending' and 'pending'.
// The network thread attempts to push data from 'sending' as fast
// as possible, whether or not updates have been created.
//
// Whenever a new delta is produced, we update the 'pending' vector.
//
// As soon as a batch of updates is sent, the 'sending' and 'pending'
// vectors are swapped.
class OutgoingWeights {
private:
    Matrix *_sending;
    Matrix *_pending;
    Matrix *_tmp;
    int64_t _id;

    vector<MPI::Request> _reqs;
public:
    OutgoingWeights(int64_t id, int numRows, int numCols) :
                    _id(id) {
        _sending = new Matrix(numRows, numCols);
        _pending = new Matrix(numRows, numCols);
        _tmp = new Matrix(numRows, numCols);
    }

    void addDelta(const NVMatrix& m) {
        m.copyToHost(*_tmp);
        _pending->add(*_tmp);
    }

    void startSend() {
        for (int i = 0; i < MPI::COMM_WORLD.Get_size(); ++i) {
            if (i == MPI::COMM_WORLD.Get_rank()) {
                continue;
            }
            // Log_Info("Sending batch... %d %d", _id, _out->getNumElements() * 4);
            _reqs.push_back(MPI::COMM_WORLD.Isend(_sending->getData(), _sending->getNumElements(), MPI::FLOAT, i, _id));
        }
    }

    bool sendDone() {
        return _reqs.empty() || MPI::Request::Testall(_reqs.size(), &_reqs[0]);
    }

    void swapPending() {
        _sending->scale(0);
        std::swap(_sending, _pending);
        _reqs.clear();
    }
};

class IncomingWeights {
private:
    MPI::Request _req;
    bool _started;
    int64_t _id;
    Matrix *_tgt;
public:
    IncomingWeights(int64_t id, Matrix* tgt) :
                    _started(false), _id(id), _tgt(tgt) {
    }

    void startRecv() {
        assert(!_started);
        _started = true;
        _req = MPI::COMM_WORLD.Irecv(_tgt->getData(), _tgt->getNumElements(), MPI::FLOAT, MPI::ANY_SOURCE, _id);
    }

    bool recvDone() {
        MPI::Status stat;
        bool done = _req.Test(stat);
        if (!done) {
            return false;
        }
        Log_Assert(stat.Get_count(MPI::FLOAT) == _tgt->getNumElements(), "Unexpected recv: %d %d %d",
                        _id, _tgt->getNumElements() * 4, stat.Get_count(MPI::FLOAT));
        return true;
    }

    void reset() {
        _started = false;
    }
};

WeightData::WeightData(int64_t id, WeightCombiner* combiner) {
    pthread_mutex_init(&sendMutex, NULL);
    pthread_mutex_init(&recvMutex, NULL);

    incReady = false;
    incCount = 0;
    this->id = id;

    incoming = NULL;
    outgoing = NULL;
    this->combiner = combiner;

    initialized = false;
}

void WeightData::initialize(int numRows, int numCols) {
    outgoing = new OutgoingWeights(id, numRows, numCols);
    recvTmp.resize(numRows, numCols);
    inc.resize(numRows, numCols);
    initialized = true;
}

bool WeightData::handleRecv() {
    if (incoming == NULL) {
        incoming = new IncomingWeights(id, &recvTmp);
        incoming->startRecv();
    }
    if (incoming->recvDone()) {
        {
            // _gpuTmp is shared across WeightData instances, but is only used from the MPI thread.
            // _gpuTmp.resize(inc);
            // _gpuTmp.copyFromHost(recvTmp);
            ScopedLock l(recvMutex);
            // inc.add(_gpuTmp);
            combiner->newGradient(recvTmp, inc);
            // inc.add(recvTmp);
            incReady = true;
        }
        incoming->reset();
        incoming->startRecv();
        return true;
    }
    return false;
}

bool WeightData::handleSend() {
    {
        ScopedLock l(sendMutex);
        if (outgoing->sendDone()) {
            outgoing->swapPending();
            outgoing->startSend();
            return true;
        }
        return false;
    }
}

NetworkManager* NetworkManager::get() {
    if (_instance != NULL) {
        return _instance;
    }

    _instance = new NetworkManager();
    return _instance;
}

NetworkManager::NetworkManager() {
    _cudaDevice = -1;
    _pause = _isPaused = false;
    _mpiThread = NULL;
    _bytesRecv = 0;
    _bytesSent = 0;
    _timeWasted = 0;
}

void NetworkManager::initialize() {
    NetworkManager* w = NetworkManager::get();
    assert(hipGetDevice(&w->_cudaDevice) == hipSuccess);
    w->_mpiThread = new FuncThread(boost::bind(&NetworkManager::_mpiThreadFn, w));
}

void NetworkManager::pauseMPI() {
    NetworkManager::get()->_pause = true;
    while (!NetworkManager::get()->_isPaused) {
        Sleep(0.001);
    }

    Log_Debug("MPI thread paused.");
}

void NetworkManager::resumeMPI() {
    NetworkManager::get()->_pause = false;
    while (NetworkManager::get()->_isPaused) {
        Sleep(0.001);
    }

    Log_Debug("MPI thread resumed.");
}

void NetworkManager::_mpiThreadFn() {
    Log_Info("Starting MPI worker thread, using CUDA device: %d", _cudaDevice);
    assert(hipSetDevice(_cudaDevice) == hipSuccess);
    cublasInit();
    while (1) {
        Sleep(0.01);
        if (_pause) {
            _isPaused = true;
            continue;
        }

        _isPaused = false;
        for (int i = 0; i < _weights.size(); ++i) {
            WeightData* w = _weights[i];
            if (!w->initialized) {
                continue;
            }

            if (w->handleRecv()) {
                _bytesRecv += w->recvTmp.getNumDataBytes();
            }
            if (w->handleSend()) {
                _bytesSent += w->recvTmp.getNumDataBytes() * (MPI::COMM_WORLD.Get_size() - 1);
            }
        }
    }
}

void NetworkManager::sendAndRecv(int64_t id, NVMatrix& gradient, NVMatrix& increment, NVMatrix& weights, int numCases) {
    TimerBlock tt(_timeWasted);

    WeightData* w = _weights[id];
    if (!w->initialized) {
        ScopedLock lw(w->sendMutex);
        ScopedLock lr(w->recvMutex);
        w->initialize(gradient.getNumRows(), gradient.getNumCols());
    }

    {
        ScopedLock l(w->sendMutex);
        w->combiner->transformGradient(gradient);
        w->outgoing->addDelta(gradient);
    }

    if (w->incReady) {
        ScopedLock l(w->recvMutex);
        assert(gradient.getNumRows() == w->inc.getNumRows());
        assert(gradient.getNumCols() == w->inc.getNumCols());

        _gpuTmp.resize(w->inc);
        _gpuTmp.copyFromHost(w->inc);
        w->combiner->newGradient(_gpuTmp, gradient);
        w->combiner->apply(weights, increment, _gpuTmp, numCases);
        w->inc.scale(0);
        w->incCount = 0;
    } else {
        w->combiner->apply(weights, increment, gradient, numCases);
    }

    PERIODIC(5,
                    Log_Info("MPI status: %.2fMB sent, %.2fMB received, %.2f seconds", _bytesSent / 1e6, _bytesRecv / 1e6, _timeWasted));
}

int64_t NetworkManager::newId(WeightCombiner* combiner) {
    int64_t id = _weights.size();
    _weights.push_back(new WeightData(id, combiner));
    return id;
}
